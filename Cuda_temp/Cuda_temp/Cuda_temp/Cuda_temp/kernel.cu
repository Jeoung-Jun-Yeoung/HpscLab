#include "hip/hip_runtime.h"
#include "kernel.cuh"


__host__ __device__
double getValue(int M, int N, int x_row, int y_col, double* List) {
	int Ind = x_row * N + y_col;
	return List[Ind];
}

__host__ __device__
int getRowInd(int M, int N, int Ind) {
	return (int)(Ind / N);
}

__host__ __device__
int getColInd(int M, int N, int Ind) {
	return (int)(Ind % N);
}

__host__ __device__
void getMulti(int M, int N, int K, int ind, double* A, double* B, double* C) {
	C[ind] = 0.;
	int x_row = getRowInd(M, K, ind);
	int y_col = getColInd(M, K, ind);

	for (int i = 0; i < N; i++) {
		double a = getValue(M, N, x_row, i, A);
		double b = getValue(N, K, i, y_col, B);

		C[ind] += a * b;
	}
}

__global__
void Kernel(int M, int N, int K, double* A, double* B, double* C) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	// blockDim.x = ���� ���� �ִ� ������ ����.
	// blockIdx.x = �����尡 ���° ���� ���ϴ���.
	// ���Ѱ��� ���� ù��° �����尡 �� �������� ���° ���������� ���ϰ�, ���ȿ��� ���°���� �����ָ� �߻��� �������� ���° ���������� �� �� �ִ�.

	// gpu������ 32, �� sm������ ������ ����

	if(id < M*K) { 
		getMulti(M, N, K, id, A, B, C);
	}
	// ������ �ּҰ����� ���� �ٸ� ������ �ϰ� �ȴ�.
}