#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define BYTE unsigned char

using namespace std;

class aes_block
{
public:
    BYTE block[16];
};

void en_file_Print(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++)
   fprintf(fp, "%02x ", b[i]);
fprintf(fp, "\n");
}


BYTE SBox[] =
{   /*0    1    2    3    4    5    6    7    8    9    a    b    c    d    e    f */
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76, /*0*/ 
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0, /*1*/
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15, /*2*/
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75, /*3*/
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84, /*4*/
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf, /*5*/
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8, /*6*/ 
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2, /*7*/
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73, /*8*/
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb, /*9*/
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79, /*a*/
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08, /*b*/
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a, /*c*/
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e, /*d*/
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf, /*e*/
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16  /*f*/
};


__device__ void AES_SubBytes(BYTE state[], BYTE sbox[]) {
int i;
for(i = 0; i < 16; i++)
    state[i] = sbox[state[i]];
}

__device__ void AES_AddRoundKey(BYTE state[], BYTE rkey[]) {
    int i;
    for(i = 0; i < 16; i++)
        state[i] ^= rkey[i];
}

__device__ void AES_ShiftRows(BYTE state[], BYTE shifttab[]) {
    BYTE h[16];
    memcpy(h, state, 16);
    int i;
    for(i = 0; i < 16; i++)
        state[i] = h[shifttab[i]];
}

__device__ void AES_MixColumns(BYTE state[], BYTE AES_xtime[]) {
    int i;
#pragma unroll
for(i = 0; i < 16; i += 4) {
    BYTE s0 = state[i + 0], s1 = state[i + 1];
    BYTE s2 = state[i + 2], s3 = state[i + 3];
    BYTE h = s0 ^ s1 ^ s2 ^ s3;
    // a + b + c + d 갈루아에서는 + 가 xor
    state[i + 0] ^= h ^ AES_xtime[s0 ^ s1];
    state[i + 1] ^= h ^ AES_xtime[s1 ^ s2];
    state[i + 2] ^= h ^ AES_xtime[s2 ^ s3];
    state[i + 3] ^= h ^ AES_xtime[s3 ^ s0];
}
}


__device__ void AES_initialization(BYTE SBox[], BYTE AES_ShiftRow[], BYTE SBox_Inv[], BYTE AES_xtime[], BYTE AES_ShiftRow_Inv[]) {
    AES_ShiftRow[0]=0;
    AES_ShiftRow[1]=5;
    AES_ShiftRow[2]=10;
    AES_ShiftRow[3]=15;
    AES_ShiftRow[4]=4;
    AES_ShiftRow[5]=9;
    AES_ShiftRow[6]=14;
    AES_ShiftRow[7]=3;
    AES_ShiftRow[8]=8;
    AES_ShiftRow[9]=13;
    AES_ShiftRow[10]=2;
    AES_ShiftRow[11]=7;
    AES_ShiftRow[12]=12;
    AES_ShiftRow[13]=1;
    AES_ShiftRow[14]=6;
    AES_ShiftRow[15]=11;
    
SBox[0] = 0x63;SBox[1] = 0x7c;SBox[2] = 0x77;SBox[3] = 0x7b;SBox[4] = 0xf2;SBox[5] = 0x6b;SBox[6] = 0x6f;SBox[7] = 0xc5;SBox[8] = 0x30;SBox[9] = 0x1;SBox[10] = 0x67;SBox[11] = 0x2b;SBox[12] = 0xfe;SBox[13] = 0xd7;SBox[14] = 0xab;SBox[15] = 0x76;
SBox[16] = 0xca;SBox[17] = 0x82;SBox[18] = 0xc9;SBox[19] = 0x7d;SBox[20] = 0xfa;SBox[21] = 0x59;SBox[22] = 0x47;SBox[23] = 0xf0;SBox[24] = 0xad;SBox[25] = 0xd4;SBox[26] = 0xa2;SBox[27] = 0xaf;SBox[28] = 0x9c;SBox[29] = 0xa4;SBox[30] = 0x72;SBox[31] = 0xc0;
SBox[32] = 0xb7;SBox[33] = 0xfd;SBox[34] = 0x93;SBox[35] = 0x26;SBox[36] = 0x36;SBox[37] = 0x3f;SBox[38] = 0xf7;SBox[39] = 0xcc;SBox[40] = 0x34;SBox[41] = 0xa5;SBox[42] = 0xe5;SBox[43] = 0xf1;SBox[44] = 0x71;SBox[45] = 0xd8;SBox[46] = 0x31;SBox[47] = 0x15;
SBox[48] = 0x4;SBox[49] = 0xc7;SBox[50] = 0x23;SBox[51] = 0xc3;SBox[52] = 0x18;SBox[53] = 0x96;SBox[54] = 0x5;SBox[55] = 0x9a;SBox[56] = 0x7;SBox[57] = 0x12;SBox[58] = 0x80;SBox[59] = 0xe2;SBox[60] = 0xeb;SBox[61] = 0x27;SBox[62] = 0xb2;SBox[63] = 0x75;
SBox[64] = 0x9;SBox[65] = 0x83;SBox[66] = 0x2c;SBox[67] = 0x1a;SBox[68] = 0x1b;SBox[69] = 0x6e;SBox[70] = 0x5a;SBox[71] = 0xa0;SBox[72] = 0x52;SBox[73] = 0x3b;SBox[74] = 0xd6;SBox[75] = 0xb3;SBox[76] = 0x29;SBox[77] = 0xe3;SBox[78] = 0x2f;SBox[79] = 0x84;
SBox[80] = 0x53;SBox[81] = 0xd1;SBox[82] = 0x0;SBox[83] = 0xed;SBox[84] = 0x20;SBox[85] = 0xfc;SBox[86] = 0xb1;SBox[87] = 0x5b;SBox[88] = 0x6a;SBox[89] = 0xcb;SBox[90] = 0xbe;SBox[91] = 0x39;SBox[92] = 0x4a;SBox[93] = 0x4c;SBox[94] = 0x58;SBox[95] = 0xcf;
SBox[96] = 0xd0;SBox[97] = 0xef;SBox[98] = 0xaa;SBox[99] = 0xfb;SBox[100] = 0x43;SBox[101] = 0x4d;SBox[102] = 0x33;SBox[103] = 0x85;SBox[104] = 0x45;SBox[105] = 0xf9;SBox[106] = 0x2;SBox[107] = 0x7f;SBox[108] = 0x50;SBox[109] = 0x3c;SBox[110] = 0x9f;SBox[111] = 0xa8;
SBox[112] = 0x51;SBox[113] = 0xa3;SBox[114] = 0x40;SBox[115] = 0x8f;SBox[116] = 0x92;SBox[117] = 0x9d;SBox[118] = 0x38;SBox[119] = 0xf5;SBox[120] = 0xbc;SBox[121] = 0xb6;SBox[122] = 0xda;SBox[123] = 0x21;SBox[124] = 0x10;SBox[125] = 0xff;SBox[126] = 0xf3;SBox[127] = 0xd2;
SBox[128] = 0xcd;SBox[129] = 0xc;SBox[130] = 0x13;SBox[131] = 0xec;SBox[132] = 0x5f;SBox[133] = 0x97;SBox[134] = 0x44;SBox[135] = 0x17;SBox[136] = 0xc4;SBox[137] = 0xa7;SBox[138] = 0x7e;SBox[139] = 0x3d;SBox[140] = 0x64;SBox[141] = 0x5d;SBox[142] = 0x19;SBox[143] = 0x73;
SBox[144] = 0x60;SBox[145] = 0x81;SBox[146] = 0x4f;SBox[147] = 0xdc;SBox[148] = 0x22;SBox[149] = 0x2a;SBox[150] = 0x90;SBox[151] = 0x88;SBox[152] = 0x46;SBox[153] = 0xee;SBox[154] = 0xb8;SBox[155] = 0x14;SBox[156] = 0xde;SBox[157] = 0x5e;SBox[158] = 0xb;SBox[159] = 0xdb;
SBox[160] = 0xe0;SBox[161] = 0x32;SBox[162] = 0x3a;SBox[163] = 0xa;SBox[164] = 0x49;SBox[165] = 0x6;SBox[166] = 0x24;SBox[167] = 0x5c;SBox[168] = 0xc2;SBox[169] = 0xd3;SBox[170] = 0xac;SBox[171] = 0x62;SBox[172] = 0x91;SBox[173] = 0x95;SBox[174] = 0xe4;SBox[175] = 0x79;
SBox[176] = 0xe7;SBox[177] = 0xc8;SBox[178] = 0x37;SBox[179] = 0x6d;SBox[180] = 0x8d;SBox[181] = 0xd5;SBox[182] = 0x4e;SBox[183] = 0xa9;SBox[184] = 0x6c;SBox[185] = 0x56;SBox[186] = 0xf4;SBox[187] = 0xea;SBox[188] = 0x65;SBox[189] = 0x7a;SBox[190] = 0xae;SBox[191] = 0x8;
SBox[192] = 0xba;SBox[193] = 0x78;SBox[194] = 0x25;SBox[195] = 0x2e;SBox[196] = 0x1c;SBox[197] = 0xa6;SBox[198] = 0xb4;SBox[199] = 0xc6;SBox[200] = 0xe8;SBox[201] = 0xdd;SBox[202] = 0x74;SBox[203] = 0x1f;SBox[204] = 0x4b;SBox[205] = 0xbd;SBox[206] = 0x8b;SBox[207] = 0x8a;
SBox[208] = 0x70;SBox[209] = 0x3e;SBox[210] = 0xb5;SBox[211] = 0x66;SBox[212] = 0x48;SBox[213] = 0x3;SBox[214] = 0xf6;SBox[215] = 0xe;SBox[216] = 0x61;SBox[217] = 0x35;SBox[218] = 0x57;SBox[219] = 0xb9;SBox[220] = 0x86;SBox[221] = 0xc1;SBox[222] = 0x1d;SBox[223] = 0x9e;
SBox[224] = 0xe1;SBox[225] = 0xf8;SBox[226] = 0x98;SBox[227] = 0x11;SBox[228] = 0x69;SBox[229] = 0xd9;SBox[230] = 0x8e;SBox[231] = 0x94;SBox[232] = 0x9b;SBox[233] = 0x1e;SBox[234] = 0x87;SBox[235] = 0xe9;SBox[236] = 0xce;SBox[237] = 0x55;SBox[238] = 0x28;SBox[239] = 0xdf;
SBox[240] = 0x8c;SBox[241] = 0xa1;SBox[242] = 0x89;SBox[243] = 0xd;SBox[244] = 0xbf;SBox[245] = 0xe6;SBox[246] = 0x42;SBox[247] = 0x68;SBox[248] = 0x41;SBox[249] = 0x99;SBox[250] = 0x2d;SBox[251] = 0xf;SBox[252] = 0xb0;SBox[253] = 0x54;SBox[254] = 0xbb; SBox[255] = 0x16;
   
    int i;
    for(i = 0; i < 256; i++){
        SBox_Inv[SBox[i]] = i;
    }
    for(i = 0; i < 16; i++)
        AES_ShiftRow_Inv[AES_ShiftRow[i]] = i;
    for(i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        // {0 * 02}
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}
 
int AES_ExpandKey(BYTE key[], int keyLen) {
    int kl = keyLen, ks, Rcon = 1, i, j;
    BYTE temp[4], temp2[4];
    switch (kl) {
        case 16: ks = 16 * (10 + 1); break;
        case 24: ks = 16 * (12 + 1); break;
        case 32: ks = 16 * (14 + 1); break;
        default: 
        printf("AES_ExpandKey: Only key lengths of 16, 24 or 32 bytes allowed!");
}
    for(i = kl; i < ks; i += 4) {
        memcpy(temp, &key[i-4], 4);
    if (i % kl == 0) {
        temp2[0] = SBox[temp[1]] ^ Rcon;
        temp2[1] = SBox[temp[2]];
        temp2[2] = SBox[temp[3]];
        temp2[3] = SBox[temp[0]];
        memcpy(temp, temp2, 4);
        if ((Rcon <<= 1) >= 256)
            Rcon ^= 0x11b;
}
    else if ((kl > 24) && (i % kl == 16)) {
        temp2[0] = SBox[temp[0]];
        temp2[1] = SBox[temp[1]];
        temp2[2] = SBox[temp[2]];
        temp2[3] = SBox[temp[3]];
        memcpy(temp, temp2, 4);
    }
    for(j = 0; j < 4; j++)
        key[i + j] = key[i + j - kl] ^ temp[j];
    }
    return ks;
}

__global__ void AES_Encrypt(aes_block aes_block_array[], BYTE key[], int keyLen, int block_number) {

    int global_thread_index = blockDim.x*blockIdx.x + threadIdx.x;
    
    __shared__ BYTE AES_ShiftRow[16];
    __shared__ BYTE SBox[256];
    __shared__ BYTE AES_ShiftRow_Inv[16];
    __shared__ BYTE SBox_Inv[256];
    __shared__ BYTE AES_xtime[256];
    //printf("blockDim.x %d blockIdx.x %d threadIdx.x %d\n",blockDim.x,blockIdx.x,threadIdx.x);
    //printf("global thread index %d \n",global_thread_index);
    //printf("block_number %d \n",block_number);
    //printf("block Dim %d \n",blockDim.x);
    
    if(threadIdx.x == 0 ){
            printf("hello from thread 0\n");
           // printf("blockDim.x %d blockIdx.x %d threadIdx.x %d\n",blockDim.x,blockIdx.x,threadIdx.x);
            //printf("global thread index %d \n",global_thread_index);
            //printf("block_number %d \n",block_number);
            //printf("block Dim %d \n",blockDim.x);
            AES_initialization(SBox, AES_ShiftRow, SBox_Inv, AES_xtime, AES_ShiftRow_Inv);
        }
        __syncthreads();

    while(global_thread_index < block_number){
    //printf("in global thread %d \n",global_thread_index);        
        BYTE block[16]; 

        for(int i=0; i<16; i++){
            block[i] = aes_block_array[global_thread_index].block[i];
        }
        int l = keyLen, i;
        AES_AddRoundKey(block, &key[0]);
        for(i = 16; i < l - 16; i += 16) {
            AES_SubBytes(block, SBox);
            AES_ShiftRows(block, AES_ShiftRow);
            AES_MixColumns(block, AES_xtime);
            AES_AddRoundKey(block, &key[i]);
        }
        AES_SubBytes(block, SBox);
        AES_ShiftRows(block, AES_ShiftRow);
        AES_AddRoundKey(block, &key[i]);

        for(int i=0; i<16; i++){
         aes_block_array[global_thread_index].block[i] = block[i];
        }
    //printf("before %d \n",global_thread_index);
    global_thread_index = global_thread_index + 256;
    //printf("after %d \n",global_thread_index);
    }
    
}

int main(int argc, char* argv[]) {

    
    ifstream ifs;
    ifs.open(argv[1], std::ifstream::binary);
    if(!ifs){
        cerr<<"Cannot open file"<<endl;
        exit(1);
    }
    // binary mode로 plain text open

    ifs.seekg(0, ios::end);
    int infileLength = ifs.tellg();
    ifs.seekg (0, ios::beg);
    cout<<"Length of input file: "<<infileLength<<endl;

    // plaintext length check


int block_number = infileLength/16 ;
// block length

int number_of_zero_pending = infileLength%16;
// empty check

printf("block_number %d number_of_zero_pending %d \n",block_number,number_of_zero_pending);
// check

aes_block* aes_block_array;

BYTE key[16 * (14 + 1)];

int keyLen = 0;
int blockLen = 16;

ifstream key_fp;
key_fp.open(argv[2]);
while(key_fp.peek()!=EOF)
{
        key_fp>>key[keyLen];
        if(key_fp.eof())
            break;
        keyLen++;
}

//key 값 담기 & key 길이 체크

cout<<keyLen<<endl;

switch (keyLen)
{
  case 16:break;
  case 24:break;
  case 32:break;
  default:printf("Key length should be 128, 192, 256bits\n"); return 0;
}

// key bit 확인

int expandKeyLen = AES_ExpandKey(key, keyLen);


if(number_of_zero_pending != 0) // 블럭이 안나눠떨어지면 1개 더만들기
    aes_block_array = new aes_block [ block_number + 1];
else
    aes_block_array = new aes_block[ block_number ];

char temp[16];

for(int i=0; i<block_number; i++){
    ifs.read(temp, 16);
    for(int j=0; j<16; j++){
        aes_block_array[i].block[j] = (unsigned char)temp[j];
    }
}
// block에 plaintext 쓰기.


if(number_of_zero_pending != 0)
{
    ifs.read(temp, number_of_zero_pending);
    for(int j=0; j<16; j++){
        aes_block_array[block_number].block[j] = (unsigned char)temp[j];
    }
    for(int j=1; j<=16-number_of_zero_pending; j++)
        aes_block_array[block_number].block[16-j] = '\0';
    block_number++;
}


hipSetDevice(0);	
hipDeviceProp_t prop;
hipGetDeviceProperties(&prop, 0);

int num_sm = prop.multiProcessorCount;

// num_sm -> gpu에서 최대치로 쓸수있는 코어 갯수 뽑아내기.

aes_block *cuda_aes_block_array;
BYTE *cuda_key;


int thrdperblock = block_number/num_sm;

// 블럭숫자를 코어로 나눈다. 즉 코어 1개가 몇개의 쓰레드를 생성할것인지 정하기.

printf("threadperblock %d\n ",thrdperblock);

if(block_number%num_sm>0)
    printf("what %d \n",block_number%num_sm);
    thrdperblock++;
// 안나눠 떨어지면 쓰레드 1개 더 생성.

if(thrdperblock>1024){
    printf("ck");
    thrdperblock = 1024;
    num_sm = block_number/1024;
    if(block_number%1024>0){
        num_sm++;
    }
}
// 최대 쓰레드 숫자를 넘어가면 코어숫자를 늘려주기.

//dim3 ThreadperBlock(thrdperblock);
//dim3 ThreadperBlock(1024);
//dim3 ThreadperBlock(512);

// thread 숫자를 256, 512, 1024 늘려가기

dim3 ThreadperBlock(256);

printf("\nThreads per block: %d\n", thrdperblock);

printf("\nthread per block dim3 %d %d %d \n",ThreadperBlock.x,ThreadperBlock.y,ThreadperBlock.z);

num_sm = 1;
// 블럭은 1개만 사용하기.


dim3 BlockperGrid(num_sm);

hipMalloc(&cuda_aes_block_array, block_number*sizeof(class aes_block));
hipMalloc(&cuda_key,16*15*sizeof(BYTE) );
// memory할당

hipMemcpy(cuda_aes_block_array, aes_block_array, block_number*sizeof(class aes_block), hipMemcpyHostToDevice);
hipMemcpy(cuda_key, key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice);
// gpu 복사 부분


//출력부분 
printf("BlockperGrid %d %d %d ThreadperBlock %d %d %d \n",BlockperGrid.x,BlockperGrid.y,BlockperGrid.z,ThreadperBlock.x,ThreadperBlock.y,ThreadperBlock.z);

hipEvent_t startEncrypt, endEncrypt;

float encrypt_time;

hipEventCreate(&startEncrypt);
hipEventCreate(&endEncrypt);

hipEventRecord(startEncrypt,0);
//ThreadperBlock
AES_Encrypt <<< BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_key, expandKeyLen, block_number);
hipDeviceSynchronize();

hipMemcpy(aes_block_array, cuda_aes_block_array, block_number*sizeof(class aes_block), hipMemcpyDeviceToHost);

hipEventRecord(endEncrypt,0);
hipEventSynchronize(endEncrypt);
hipEventElapsedTime(&encrypt_time, startEncrypt, endEncrypt);

hipError_t err;
err = hipGetLastError(); // `cudaGetLastError` will return the error from above.
// thread 숫자를 256개 이상 쓰면 waring : Too Many Resources Requested for Launch
if (err != hipSuccess)
{
  printf("Error: %s\n", hipGetErrorString(err));
}


// 결과 기록용
printf("\nEncrypt time %f ms \n",encrypt_time);

FILE* timefp = fopen("text50kb_thread256.txt","a");
fprintf(timefp,"16 Encrypt time %f ms \n",encrypt_time);
fclose(timefp);

FILE* en_fp;

en_fp = fopen(argv[3], "wb");

for(int i=0; i<block_number-1; i++){
    en_file_Print(aes_block_array[i].block, blockLen, en_fp);
}
if(number_of_zero_pending == 0)
    en_file_Print(aes_block_array[block_number-1].block, blockLen, en_fp);
else 
    en_file_Print(aes_block_array[block_number-1].block, blockLen, en_fp);

fclose(en_fp);

    return 0;
}