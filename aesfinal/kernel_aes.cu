#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define BYTE unsigned char

class aes_block
{
public:
    BYTE block[16];
};

__device__ add_roundkey(){

}

__device__ sub_byte(BYTE state [], BYTE shift_bit[]){

}
__device__ shift_row(){

}
__device__ mix_column(){

}

__global__ void encrypt(){

}


int main(int argc, char* argv[]){
    FILE* plaintext_fp;

    plaintext_fp = fopen(argv[1], "rb");

    if(plaintext_fp == NULL){
        printf("the file to encypt does not exist\n");
        return 0;
    }
    // 평문이 담긴 txt 파일 오픈.

    int plaintext_size;

    fseek(plaintext_fp, 0 , SEEK_END);
    plaintext_size = ftell(plaintext_fp);

    // 평문 text size 측정.

    fseek(fp,0,SEEK_SET);

    // 평문 포인터 다시 처음으로.

    printf("plaintext length %d \n", plaintext_size); // print to plaintext

    int block_number = plaintext_size / 16;

    int empty_space_in_block = plaintext_size % 16;

    printf("block_number %d empty_space_in_block %d \n",block_number, empty_space_in_block);
    // print

    if(empty_space_in_block != 0){
        aes_block_array = new aes_block [block_number + 1];
    }
    else{
        aes_block_array = new aes_block [block_number];
    }
    // block asign
    char temp_plaintext[16];

    for(int i = 0; i < block_number;){
        fgets(temp_plaintext,16,plaintext_fp);
        for(int j = 0; j < 16; j++){
            aes_block_array[i].block[j] = (unsigned char) temp_plaintext[j];
        }
    }

    if(empty_space_in_block != 0){
        fgets(temp_plaintext,empty_space_in_block,plaintext_fp);
        int start = 0;
        for(int j = 0; j < 16; j++){
            aes_block_array[block_number].block[j] = (unsigned char)temp_plaintext[j];
            start = j + 1;
        }
        // 일단 남은 내용들을 채워준다.
        for(int k = start; k < 16; k ++)
            aes_block_array[block_number].block[k] = '\0';
        block_number++
        //이후 빈공간을 \0으로 채우기
    }

    // plaint text -> aes_block_array

    /*
    plaintext : abcdefghijklmnopqrstu

    aes_block_array[0].block[0]~[16]
    {a,b,c,d,e,f,g,h,i,j,k,l,m,n,o}
    aes_block_array[1].block[0]~[16]
    {p,q,r,s,t,u,\0,\0,\0,\0,\0,\0,\0,\0,\0,\0,\0}
    
    */

    
    FILE* key_fp;

    key_fp = fopen(argv[2],"r");
    char read_key[16];

    fgets(read_key, 16, key_fp);

    /*
    key_expansion
    */







}