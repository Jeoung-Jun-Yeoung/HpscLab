#include "encrypt.cuh"
#include <cstdio>
#include <cstdlib>
#include <cassert>

//ANSIX923

//PKCS7 padding
// "0123456789ABCDEF"
// "\x16\x16\x16....."
// "123" -> 16
// "123\x13\x13\x13...."

// "010203...." -> [0x01, 0x02, 0x03, ..]


int main(int argc, char *argv[])
{
    FILE *plaintext_fp;
    FILE *key_fp;

    plaintext_fp = fopen(argv[1], "rb");

    if (plaintext_fp == NULL)
    {
        printf("the file to encypt does not exist\n");
        return 0;
    }
    // 평문이 담긴 txt 파일 오픈.
    // Hex string

    int plaintext_size;

    fseek(plaintext_fp, 0, SEEK_END);
    plaintext_size = ftell(plaintext_fp);


    // 평문 text size 측정.

    fseek(plaintext_fp, 0, SEEK_SET);

    // 평문 포인터 다시 처음으로.
    int pt_byte_size = plaintext_size/2;
    printf("plaintext length %d \n", plaintext_size); // print to plaintext
    
    //fgets(temp_plain_text, plaintext_size + 1, plaintext_fp);
    BYTE *plain_text = (BYTE *)malloc(sizeof(BYTE) *  pt_byte_size);
    for(int i = 0; i < pt_byte_size; i++){
        char buf[3] = {0, 0, 0};
        fread(buf, 2, 1, plaintext_fp);
        //fseek(plaintext_fp, 2 , SEEK_CUR);
        plain_text[i] = strtol(buf, NULL, 16);
    }

    //for(int i=0; i<pt_byte_size; i++){
      //  printf("%02x ", plain_text[i]);
    //}
    //printf("\n");

    key_fp = fopen(argv[2], "rb");

    if (key_fp == NULL)
    {
        printf("key file does not exist\n");
        return 0;
    }
    // key txt 파일 오픈.

    int key_size;

    fseek(key_fp, 0, SEEK_END);
    key_size = ftell(key_fp);

    // 평문 text size 측정.

    fseek(key_fp, 0, SEEK_SET);

    printf("key length %d \n", key_size);
    assert(key_size == 32);
    key_size = 16;
    BYTE key[16];

    for(int i = 0; i < key_size; i++){
        char buf[3] = {0, 0, 0};
        fread(buf, 2, 1, key_fp);
        key[i] = strtol(buf, NULL, 16);
    }


    BYTE key_expand[16 * (ROUNDS + 1)];

    AES_ExpandKey(key, key_expand);

    //000102030405060708090a0b0c0d0e0f
    
    BYTE *device_p_text;
    BYTE *device_key;

    hipMalloc(&device_p_text, pt_byte_size);
    hipMalloc(&device_key, 16 * (ROUNDS + 1));
    hipMemcpy(device_p_text,plain_text,pt_byte_size,hipMemcpyHostToDevice);
    hipMemcpy(device_key, key_expand , 16 * (ROUNDS + 1),hipMemcpyHostToDevice);

    int thread = atoi(argv[3]);
    printf("threads %d \n", thread);
    dim3 ThreadperBlock(thread);
    int sm = 1;
    dim3 BlokcperGrid(sm);
    hipEvent_t startEncrypt, endEncrypt;

    float encrypt_time;

    printf("pt_byte: %d \n", pt_byte_size);
    assert (pt_byte_size%16 == 0);

    hipEventCreate(&startEncrypt);
    hipEventCreate(&endEncrypt);
    hipEventRecord(startEncrypt,0);

    Cipher<<<BlokcperGrid,ThreadperBlock>>>(device_p_text, pt_byte_size, device_key, thread);
    hipMemcpy(plain_text, device_p_text, pt_byte_size,hipMemcpyDeviceToHost);
    
    hipEventRecord(endEncrypt,0);
    hipEventSynchronize(endEncrypt);
    hipEventElapsedTime(&encrypt_time, startEncrypt, endEncrypt);



    hipError_t err;
    err = hipGetLastError(); // `hipGetLastError` will return the error from above.
    // thread 숫자를 256개 이상 쓰면 waring : Too Many Resources Requested for Launch
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }   


    // 결과 기록용
    printf("\nEncrypt time %f ms \n",encrypt_time);

    FILE* timefp = fopen("5000kb_measure.txt","a");
    fprintf(timefp,"thread %d ",thread);
    fprintf(timefp,"Encrypt time %f ms \n",encrypt_time);
    fclose(timefp);
    // 2 4 8 16 32 64 128 256 512 1024
   
    //for(int i=0; i<pt_byte_size; i++){
    //   printf("%02x ", plain_text[i]);
    //}
    
    //printf("\n");
    
}