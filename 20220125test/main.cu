#include <stdio.h>
#include <stdlib.h>

#include "kernel.cuh"

int main() {

	hipSetDevice(0);

	int M, N, K;
	M = N = 3;
	K = 1;

	double *A, *B, *C , *d_A, *d_B, *d_C;

	A = (double*)malloc(sizeof(double) * M * N);
	B = (double*)malloc(sizeof(double) * N * K);
	C = (double*)malloc(sizeof(double) * M * K);

	hipMalloc(&d_A, sizeof(double) * M * N);
	hipMalloc(&d_B, sizeof(double) * N * K);
	hipMalloc(&d_C, sizeof(double) * M * K);


	for (int i = 0; i < M * N; i++) {
		A[i] = 1.;
	}
	for (int i = 0; i < N * K; i++) {
		B[i] = 1.;
	}
	for (int i = 0; i < M * K; i++) {
		C[i] = 1.;
	}

	hipMemcpy(d_A, A, sizeof(double) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(double) * N * K, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(double) * M * K, hipMemcpyHostToDevice);


	hipDeviceProp_t devProp;

	hipGetDeviceProperties(&devProp, 0);
	int nThreads = (int)(devProp.maxThreadsPerBlock / 4);

	printf("nthread %d\n",nThreads);
	//최대로 쓰나 4개를 쓰나 비슷하다.
	int nBlocks = 65535;
	printf("nBlocks %d\n",nBlocks);
	//관행

	int ind = 0;

	printf("be\n");
		while (true)
	{
		for (int i = 0; i < K; i++) {
			printf("%f \t", C[i]);
			ind += 1;
		}
		printf("\n");
		if (M * K <= ind) {
			break;
		}
	}

	Kernel<<<nBlocks, nThreads>>> (M, N, K, d_A, d_B, d_C);

	hipMemcpy(C, d_C, sizeof(double) * M * K, hipMemcpyDeviceToHost);



	ind = 0;

	while (true)
	{
		for (int i = 0; i < K; i++) {
			printf("%f \t", C[i]);
			ind += 1;
		}
		printf("\n");
		if (M * K <= ind) {
			break;
		}
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(A);
	free(B);
	free(C);

	return 0;
}