#include "hip/hip_runtime.h"
#include "kernel.cuh"


__host__ __device__
double getValue(int M, int N, int x_row, int y_col, double* List) {
	int Ind = x_row * N + y_col;
	return List[Ind];
}

__host__ __device__
int getRowInd(int M, int N, int Ind) {
	return (int)(Ind / N);
}

__host__ __device__
int getColInd(int M, int N, int Ind) {
	return (int)(Ind % N);
}

__host__ __device__
void getMulti(int M, int N, int K, int ind, double* A, double* B, double* C) {
	C[ind] = 0.;
	int x_row = getRowInd(M, K, ind);
	int y_col = getColInd(M, K, ind);

	for (int i = 0; i < N; i++) {
		double a = getValue(M, N, x_row, i, A);
		double b = getValue(N, K, i, y_col, B);

		C[ind] += a * b;
	}
}

__global__
void Kernel(int M, int N, int K, double* A, double* B, double* C) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	// blockDim.x = 블럭이 갖고 있는 쓰레드 숫자.
	// blockIdx.x = 쓰레드가 몇번째 블럭에 속하는지.
	// 곱한값은 블럭의 첫번째 쓰레드가 총 쓰레드중 몇번째 스레드인지 구하고, 블럭안에서 몇번째인지 더해주면 발생된 쓰레드중 몇번째 쓰레드인지 알 수 있다.

	// gpu에서는 32, 즉 sm단위로 쓰레드 생성

	if(id < M*K) {
		getMulti(M, N, K, id, A, B, C);
	}
	// 고유의 주소값으로 각각 다른 동작을 하게 된다.
}