
#include <hip/hip_runtime.h>
#include <stdio.h>


int main (){


	int count;

	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);

	for(int i = 0; i < count; i++){
		hipGetDeviceProperties(&prop, i);
		printf("%s \n",prop.name);
	}

	// 그래픽 카드 모델명 출력 확인.
	
}