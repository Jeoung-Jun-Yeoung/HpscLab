#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <iostream>


__global__ void kernel(int a, int b, int* c){
	*c = a + b;
}

int main(void){

	int c;
	
	int a;
	int b;

	int* dev_c;


	a = 1;

	b = 2;
	
	hipMalloc((void**) &dev_c, sizeof(int));

	printf("asdasd c %d\n",*dev_c);

	//printf("befored c %d \n",c);

	kernel<<<1,1>>>(a, b, dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

	printf("after c = %d\n",c);
	
	hipFree(dev_c);

	return 0;
}
