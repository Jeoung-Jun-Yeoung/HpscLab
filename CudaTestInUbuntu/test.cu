#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"

__global__ void add(int a, int b, int* c){
	*c = a + b;
}

int main(void){

	int c;
	
	int* a;
	int* b;

	int *dev_c = (int*)malloc(sizeof(int));

	a = (int*)malloc(sizeof(int));
	b = (int*)malloc(sizeof(int));
	
	hipMalloc( &dev_c, sizeof(int));

	add<<<1,1>>>(2, 7, dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

	printf("2 + 7 = %d\n",c);
	
	hipFree(dev_c);

	return 0;
}