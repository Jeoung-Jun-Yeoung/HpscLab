#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <iostream>


__global__ void kernel(int a, int b, int* c){
	*c = a + b;
}

int main(void){

	int c;
	
	int* dev_c;

	
	hipMalloc((void**) &dev_c, sizeof(int));

	printf("before c %d\n",*dev_c);

	//printf("befored c %d \n",c);

	kernel<<<1,1>>>(1, 2,dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

	printf("after c = %d\n",c);
	
	
	hipFree(dev_c);

	return 0;
}
