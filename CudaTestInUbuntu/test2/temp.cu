#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>



__global__ void kernel(int a, int b, int* c){
	*c = a + b;
}

int main(void){

	int c;
	
	int* dev_c;

	
	hipMalloc((void**) &dev_c, sizeof(int));


	kernel<<<1,1>>>(1, 2,dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

	printf("after c = %d\n",c);
	
	
	hipFree(dev_c);

	return 0;
}
